#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#define CUBQL_GPU_BUILDER_IMPLEMENTATION 1
#include "cuBQL/bvh.h"

#include "cuBQL/CUDAArray.h"
#include "testing/helper.h"

namespace testing {

  typedef cuBQL::box3fa box_t;

  void usage(const std::string &error = "")
  {
    if (!error.empty()) {
      std::cerr << error << "\n\n";
    }
    std::cout << "./buildPerf_points dataPoints.dat\n\n";
    exit(error.empty()?0:1);
  }

  __global__ void makeBoxes(box_t *boxes, float3 *points, int numPoints)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numPoints) return;
    float3 point = points[tid];
    boxes[tid].lower = point;
    boxes[tid].upper = point;
  }

  void buildPerf(const std::vector<float3> &h_dataPoints,
                 int maxLeafSize,
                 float numSecsAvg)
  {
    int numPrims = h_dataPoints.size();
    cuBQL::CUDAArray<float3> dataPoints;
    dataPoints.upload(h_dataPoints);
    cuBQL::CUDAArray<box_t> boxes(dataPoints.size());
    {
      int bs = 256;
      int nb = divRoundUp((int)dataPoints.size(),bs);
      makeBoxes<<<nb,bs>>>(boxes.data(),dataPoints.data(),(int)dataPoints.size());
    };
    
    cuBQL::BinaryBVH bvh;
    cuBQL::gpuBuilder(bvh,boxes.data(),boxes.size(),maxLeafSize);

    double t0 = getCurrentTime();
    int thisRunSize = 1;
    while (true) {
      double t0 = getCurrentTime();
      for (int i=0;i<thisRunSize;i++) {
        cuBQL::free(bvh);
        cuBQL::gpuBuilder(bvh,boxes.data(),boxes.size(),maxLeafSize);
      }
      double t1 = getCurrentTime();
      if ((t1 - t0) > numSecsAvg) {
        double buildsPerSecond = thisRunSize / (t1-t0);
        double primsPerSecond = (thisRunSize*(double)numPrims) / (t1-t0);
        std::cout << "done. re-built BVH " << thisRunSize << " times in " << (t1-t0) << " secs, that's " << prettyDouble(buildsPerSecond) << " builds per second (or " << prettyDouble(1.f/buildsPerSecond) << "s/build); or " << prettyDouble(primsPerSecond) << "prims/sec" << std::endl;
        std::cout << "BPS " << buildsPerSecond << std::endl;
        std::cout << "PPS " << primsPerSecond << std::endl;
        break;
      }
      thisRunSize *= 2;
    }
    cuBQL::free(bvh);
  }
}

using namespace testing;

int main(int ac, char **av)
{
  int maxLeafSize = 8;
  float numSecsAvg = 5.f;
  std::vector<std::string> fileNames;
  for (int i=1;i<ac;i++) {
    const std::string arg = av[i];
    if (av[i][0] != '-')
      fileNames.push_back(arg);
    else if (arg == "-mls" || arg == "-ls")
      maxLeafSize = std::stoi(av[++i]);
    else if (arg == "-ns")
      numSecsAvg = std::stof(av[++i]);
    else
      usage("unknown cmd-line argument '"+arg+"'");
    }
    if (fileNames.size() != 1)
      usage("unexpected number of data file names");
    std::vector<float3> dataPoints  = loadData<float3>(fileNames[0]);

    testing::buildPerf(dataPoints,maxLeafSize,numSecsAvg);
    return 0;
  }
