#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#define CUBQL_GPU_BUILDER_IMPLEMENTATION 1
#include "cuBQL/bvh.h"
#include "cuBQL/fcp.h"

#include "cuBQL/CUDAArray.h"
#include "testing/helper.h"

namespace testing {

  using box_t = cuBQL::box3f;

  void usage(const std::string &error = "")
  {
    if (!error.empty()) {
      std::cerr << error << "\n\n";
    }
    std::cout << "./cuBQL_fcp dataPoints.dat queryPoints.dat\n\n";
    exit(error.empty()?0:1);
  }

  __global__ void makeBoxes(box_t *boxes, float3 *points, int numPoints)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numPoints) return;
    float3 point = points[tid];
    boxes[tid].lower = point;
    boxes[tid].upper = point;
  }

  __global__ void resetResults(int *results, int numQueries)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numQueries) return;
    results[tid] = -1;
  }

  __global__ void runFCP(int *results,
                         BinaryBVH bvh,
                         const float3 *dataPoints,
                         const float3 *queries,
                         int numQueries)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numQueries) return;

    results[tid] = cuBQL::fcp(bvh,dataPoints,queries[tid]);
  }
  
  void testFCP(const std::vector<float3> &h_dataPoints,
               const std::vector<float3> &h_queryPoints,
               int maxLeafSize,
               float maxTimeThreshold = 10.f
               )
  {
    cuBQL::CUDAArray<float3> dataPoints;
    dataPoints.upload(h_dataPoints);
    cuBQL::CUDAArray<box_t> boxes(dataPoints.size());
    {
      int bs = 256;
      int nb = divRoundUp((int)dataPoints.size(),bs);
      makeBoxes<<<nb,bs>>>(boxes.data(),dataPoints.data(),(int)dataPoints.size());
    };
    
    cuBQL::BinaryBVH bvh;
    cuBQL::gpuBuilder(bvh,boxes.data(),boxes.size(),maxLeafSize);

    cuBQL::CUDAArray<float3> queryPoints;
    queryPoints.upload(h_queryPoints);
    
    int numQueries = queryPoints.size();
    cuBQL::CUDAArray<int> closest(numQueries);

    int numPerRun = 1;
    while (true) {
      CUBQL_CUDA_SYNC_CHECK();
      double t0 = getCurrentTime();
      for (int i=0;i<numPerRun;i++) {
        resetResults<<<divRoundUp(numQueries,128),128>>>(closest.data(),numQueries);
        runFCP<<<divRoundUp(numQueries,128),128>>>
          (closest.data(),
           bvh,
           dataPoints.data(),
           queryPoints.data(),
           numQueries);
        CUBQL_CUDA_SYNC_CHECK();
      }
      double t1 = getCurrentTime();
      std::cout << "done " << numPerRun
                << " queries in " << prettyDouble(t1-t0) << "s, that's "
                << prettyDouble((t1-t0)/numPerRun) << "s query" << std::endl;
      if ((t1 - t0) > maxTimeThreshold)
        break;
      numPerRun*=2;
    };
  }
}

using namespace testing;

int main(int ac, char **av)
{
  int maxLeafSize = 8;
  std::vector<std::string> fileNames;
  for (int i=1;i<ac;i++) {
    const std::string arg = av[i];
    if (av[i][0] != '-')
      fileNames.push_back(arg);
      else if (arg == "-mls" || arg == "-ls")
        maxLeafSize = std::stoi(av[++i]);
      else
        usage("unknown cmd-line argument '"+arg+"'");
  }
  if (fileNames.size() != 2)
    usage("unexpected number of data file names");
  std::vector<float3> dataPoints  = loadData<float3>(fileNames[0]);
  std::vector<float3> queryPoints = loadData<float3>(fileNames[1]);
  
  testing::testFCP(dataPoints,queryPoints,maxLeafSize);
  return 0;
}
