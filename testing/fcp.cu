#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "cuBQL/bvh.h"
#include "cuBQL/DevMem.h"
#include <fstream>
#include <vector>

#define CUBQL_GPU_BUILDER_IMPLEMENTATION 1
#include "cuBQL/gpu_builder.h"


namespace testing {
  
  using cubql::divRoundUp;
  
  template<typename T>
  std::vector<T> loadData(const std::string &fileName)
  {
    std::ifstream in(fileName.c_str(),std::ios::binary);
    size_t count;
    in.read((char*)&count,sizeof(count));

    std::vector<T> data(count);
    in.read((char*)data.data(),count*sizeof(T));
    return data;
  }

  void usage(const std::string &error = "")
  {
    if (!error.empty()) {
      std::cerr << error << "\n\n";
    }
    std::cout << "./cuBQL_fcp dataPoints.dat queryPoints.dat\n\n";
    exit(error.empty()?0:1);
  }

  __global__ void makeBoxes(box3f *boxes, float3 *points, int numPoints)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numPoints) return;
    float3 point = points[tid];
    boxes[tid] = { point,point };
  }

  void testFCP(const std::vector<float3> &dataPoints,
               const std::vector<float3> &queryPoints)
  {
    cubql::CUDAArray<box3f> boxes(dataPoints.size());
    {
      int bs = 256;
      int nb = divRoundUp((int)dataPoints.size(),bs);
      makeBoxes<<<nb,bs>>>(boxes.data(),dataPoints.data(),(int)dataPoints.size());
    };
    
    cubql::BinaryBVH bvh;
    cubql::gpuBuilder(bvh,boxes.data(),boxes.size(),maxLeafSize);
  }
}

using namespace testing;

int main(int ac, char **av)
{
  int maxLeafSize = 8;
  std::vector<std::string> fileNames;
  for (int i=1;i<ac;i++) {
    const std::string arg = av[i];
    if (av[i][0] != '-')
      fileNames.push_back(arg);
      else if (arg == "-mls" || arg == "-ls")
        maxLeafSize = std::stoi(av[++i]);
      else
        usage("unknown cmd-line argument '"+arg+"'");
    }
    if (fileNames.size() != 2)
      usage("unexpected number of data file names");
    std::vector<float3> dataPoints  = loadData<float3>(fileNames[0]);
    std::vector<float3> queryPoints = loadData<float3>(fileNames[1]);

    testing::testFCP(dataPoints,queryPoints,,maxLeafSize);
    return 0;
  }
